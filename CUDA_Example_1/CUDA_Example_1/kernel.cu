﻿/*
CUDA Programs
By J RAJAN
For Learners
Title: Addition of 2 Arrays in Parallel
*/

#include "hip/hip_runtime.h"


#include <stdio.h>

#define MAX 100 //Maximum length of array

__global__ void add(int* a, int* b, int* c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    int N = 10; //Actual defined size of array
    int a[MAX], b[MAX], c[MAX];
    int* d_a, * d_b, * d_c;

    //Memory allocation on GPU
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
        b[i] = i * i; //squaring for second array
    }

    //Both arrays copied to GPU memory 
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add <<<1, N>>> (d_a, d_b, d_c); //Kernel Function for addition. N number of parallel computations

    //Send data back to host memory
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost); 

    //Display 
    for (int i = 0; i < N; i++) 
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    //Deallocate memory on GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
